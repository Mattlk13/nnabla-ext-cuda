#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cudnn/cudnn.hpp>
#include <nbla/cuda/cudnn/function/instance_normalization.hpp>
#include <nbla/variable.hpp>

namespace nbla {

inline void dump_buffer(const float *ptr, const int size, string message) {
  std::cout << message << ": ";
  for (int i = 0; i < size; i++) {
    std::cout << ptr[i] << ", ";
  }
  std::cout << std::endl;
}

inline void dump_data_buffer(Variable *var, const int size, string message) {
  hipDeviceSynchronize();
  nbla::Context cpu_ctx{{"cpu:float"}, "CpuCachedArray", "0"};
  const float *ptr = var->get_data_pointer<float>(cpu_ctx);
  dump_buffer(ptr, size, message);
}

inline void dump_grad_buffer(Variable *var, const int size, string message) {
  hipDeviceSynchronize();
  nbla::Context cpu_ctx{{"cpu:float"}, "CpuCachedArray", "0"};
  const float *ptr = var->get_grad_pointer<float>(cpu_ctx);
  dump_buffer(ptr, size, message);
}

#define DRV_BN_T() get_dtype_by_cudnn_data_type(derived_bn_dtype_)

template <typename T>
void InstanceNormalizationCudaCudnn<T>::setup_impl(const Variables &inputs,
                                                   const Variables &outputs) {
  InstanceNormalizationCuda<T>::setup_impl(inputs, outputs);
  cuda_set_device(this->device_);

  if (outputs.size() == 3) {
    // [WORKAROUND]
    // To use saved mean and variance and to propagate mean and variance
    // gradient are not supported with cuDNN.
    // Because cuDNN's backward interface is different from NNabla's one.
    // So Fall back to CUDA implementation if outputs.size() == 3
    // TODO: Change saved variance to inverse variance like cuDNN
    this->fall_back_func_ = make_shared<InstanceNormalizationCuda<T>>(
        this->ctx_, this->channel_axis_, this->batch_axis_, this->eps_,
        this->no_scale_, this->no_bias_);
    this->fall_back_func_->setup(inputs, outputs);
    return;
  }

  cudnn_handle_ = SingletonManager::get<CudnnHandleManager>()->handle(device_);

  channel_last_ =
      this->channel_axis_ == inputs[0]->ndim() - 1 && inputs[0]->ndim() != 2;

  if (channel_last_) {
    const int c = this->channel_axis_;
    const auto ndim = inputs[0]->ndim();

    // Pre-transpose: [b, h, w, c] -> [b, c, h, w]
    vector<int> pre_transpose_shape;
    for (int i = 0; i < this->batch_axis_.size(); i++) {
      pre_transpose_shape.push_back(i);
    }
    pre_transpose_shape.push_back(c);
    for (int i = this->batch_axis_.size(); i < ndim - 1; i++) {
      pre_transpose_shape.push_back(i);
    }
    pre_transpose_ = create_Transpose(this->ctx_, pre_transpose_shape);
    pre_transpose_->setup({inputs[0]}, {&pre_adaptor_});

    // Post-transpose: [b, c, h, w] -> [b, h, w, c]
    vector<int> post_transpose_shape;
    for (int i = 0; i < this->batch_axis_.size(); i++) {
      post_transpose_shape.push_back(i);
    }
    for (int i = this->batch_axis_.size(); i < ndim - 1; i++) {
      post_transpose_shape.push_back(i + 1);
    }
    post_transpose_shape.push_back(this->batch_axis_.size());
    post_transpose_ = create_Transpose(this->ctx_, post_transpose_shape);
    post_adaptor_.reshape(pre_adaptor_.shape(), true);
    post_transpose_->setup({&post_adaptor_}, {outputs[0]});

    reduction_size_ = pre_adaptor_.size(this->batch_axis_.size() + 1);
    outer_size_ = pre_adaptor_.size() / reduction_size_;
  } else {
    reduction_size_ = inputs[0]->size(this->channel_axis_ + 1);
    outer_size_ = inputs[0]->size() / reduction_size_;
  }

  // [N, C, H, W] -> [1, N*C, H*W, 1]
  int N = 1;
  int C = outer_size_;
  int H = reduction_size_;
  int W = 1;

  mode_ = HIPDNN_BATCHNORM_SPATIAL;
  NBLA_CUDNN_CHECK(
      hipdnnSetTensor4dDescriptor(input_desc_.desc, HIPDNN_TENSOR_NCHW,
                                 cudnn_data_type<T>::type(), N, C, H, W));
  NBLA_CUDNN_CHECK(
      hipdnnSetTensor4dDescriptor(output_desc_.desc, HIPDNN_TENSOR_NCHW,
                                 cudnn_data_type<T>::type(), N, C, H, W));

  // Get BN data type.
  NBLA_CUDNN_CHECK(hipdnnDeriveBNTensorDescriptor(
      bn_scale_bias_mean_var_desc_.desc, input_desc_.desc, mode_));
  int n, c, h, w, sn, sc, sh, sw; // garbage
  NBLA_CUDNN_CHECK(hipdnnGetTensor4dDescriptor(bn_scale_bias_mean_var_desc_.desc,
                                              &derived_bn_dtype_, &n, &c, &h,
                                              &w, &sn, &sc, &sh, &sw));

  b_idx_ = 1;
  g_idx_ = this->no_bias_ ? 1 : 2;

  mean_.reshape({outer_size_}, true);
  var_.reshape({outer_size_}, true);
  beta_dummy_.reshape({outer_size_}, true);
  gamma_dummy_.reshape({outer_size_}, true);
}

template <typename T>
void InstanceNormalizationCudaCudnn<T>::forward_impl(const Variables &inputs,
                                                     const Variables &outputs) {
  cuda_set_device(this->device_);
  if (channel_last_) {
    pre_transpose_->forward({inputs[0]}, {&pre_adaptor_});

    auto in_cf_in = inputs;
    in_cf_in[0] = &pre_adaptor_;
    auto in_cf_out = outputs;
    in_cf_out[0] = &post_adaptor_;

    forward_channel_first(in_cf_in, in_cf_out);

    post_transpose_->forward({&post_adaptor_}, {outputs[0]});
  } else {
    forward_channel_first(inputs, outputs);
  }
}

template <typename T>
void InstanceNormalizationCudaCudnn<T>::forward_channel_first(
    const Variables &inputs, const Variables &outputs) {
  InstanceNormalizationCuda<T>::forward_channel_first(inputs, outputs);
  return;

  Variable *batch_mean = &this->mean_;
  Variable *batch_var = &this->var_;

  // Inputs
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);

  // dummy beta, gamma variables
  const auto param_shape = this->mean_.shape();
  if (this->no_bias_) {
    beta_dummy_.reshape(param_shape, true);
    beta_dummy_.data()->zero();
  }
  if (this->no_scale_) {
    gamma_dummy_.reshape(param_shape, true);
    gamma_dummy_.data()->fill(1.);
  }

  const void *beta =
      this->no_bias_
          ? beta_dummy_.data()->get(DRV_BN_T(), this->ctx_)->const_pointer()
          : inputs[this->b_idx_]
                ->data()
                ->get(DRV_BN_T(), this->ctx_)
                ->const_pointer();

  const void *gamma =
      this->no_scale_
          ? gamma_dummy_.data()->get(DRV_BN_T(), this->ctx_)->const_pointer()
          : inputs[this->g_idx_]
                ->data()
                ->get(DRV_BN_T(), this->ctx_)
                ->const_pointer();

  // Output
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  void *m = batch_mean->data()
                ->cast(DRV_BN_T(), this->ctx_, true)
                ->pointer(); // batch mean
  void *v = batch_var->data()
                ->cast(DRV_BN_T(), this->ctx_, true)
                ->pointer(); // batch var
  // Running mean and variance.
  void *rm = nullptr;
  void *rv = nullptr;

  auto a = get_cudnn_scalar_arg<T>(1);
  auto b = get_cudnn_scalar_arg<T>(0);
  double eps = std::max((double)this->eps_, HIPDNN_BN_MIN_EPSILON);
  NBLA_CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
      cudnn_handle_, mode_, &a, &b, input_desc_.desc, x, output_desc_.desc, y,
      bn_scale_bias_mean_var_desc_.desc, gamma, beta, 0, rm, rv, eps, m, v));
}

template <typename T>
void InstanceNormalizationCudaCudnn<T>::backward_impl(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  if (!(propagate_down[0] || (inputs.size() > 1 && propagate_down[1]) ||
        (inputs.size() > 2 && propagate_down[2]))) {
    return;
  }
  cuda_set_device(this->device_);

  if (channel_last_) {
    auto in_cf_in = inputs;
    in_cf_in[0] = &pre_adaptor_;
    auto in_cf_out = outputs;
    in_cf_out[0] = &post_adaptor_;

    post_transpose_->backward({&post_adaptor_}, {outputs[0]}, {true}, {false});

    auto in_cf_accum = accum;
    in_cf_accum[0] = false;
    backward_channel_first(in_cf_in, in_cf_out, propagate_down, in_cf_accum);

    pre_transpose_->backward({inputs[0]}, {&pre_adaptor_}, {propagate_down[0]},
                             {accum[0]});
  } else {
    backward_channel_first(inputs, outputs, propagate_down, accum);
  }
}

template <typename T>
void InstanceNormalizationCudaCudnn<T>::backward_channel_first(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  InstanceNormalizationCuda<T>::backward_channel_first(inputs, outputs,
                                                       propagate_down, accum);
  return;

  const bool pd_beta = !this->no_bias_ && propagate_down[this->b_idx_];
  const bool pd_gamma = !this->no_scale_ && propagate_down[this->g_idx_];

  const bool accum_beta = !this->no_bias_ && accum[this->b_idx_];
  const bool accum_gamma = !this->no_scale_ && accum[this->g_idx_];

  // Check whether it outputs batch mean/var.
  Variable *batch_mean = &this->mean_;
  Variable *batch_var = &this->var_;
  // Common inputs wrt. gradient.
  const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
  const void *m =
      batch_mean->data()->get(DRV_BN_T(), this->ctx_)->const_pointer();
  const void *v =
      batch_var->data()->get(DRV_BN_T(), this->ctx_)->const_pointer();
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);

  auto a_data = get_cudnn_scalar_arg<T>(propagate_down[0] ? 1 : 0);
  auto b_data = get_cudnn_scalar_arg<T>(accum[0] && propagate_down[0] ? 1 : 0);
  auto a_param = get_cudnn_scalar_arg<T>((pd_beta || pd_gamma) ? 1 : 0);
  auto b_param = get_cudnn_scalar_arg<T>(
      ((pd_beta && accum_beta) || (pd_gamma && accum_gamma)) ? 1 : 0);

  size_t prop_down_workspace_size = 0;
  if (!propagate_down[0]) {
    prop_down_workspace_size = std::max(
        prop_down_workspace_size, inputs[0]->size() * sizeof_dtype(DRV_BN_T()));
  }
  if (!pd_beta || !pd_gamma) {
    prop_down_workspace_size = std::max(prop_down_workspace_size,
                                        outer_size_ * sizeof_dtype(DRV_BN_T()));
  }
  void *prop_down_buf = nullptr;
  NdArray prop_down_workspace;
  if (prop_down_workspace_size) {
    prop_down_workspace.reshape({static_cast<Size_t>(prop_down_workspace_size)},
                                true);
    prop_down_buf = prop_down_workspace.cast(dtypes::BYTE, this->ctx_, true)
                        ->pointer<void>();
  }

  Tc *dx = propagate_down[0]
               ? inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, !accum[0])
               : (Tc *)prop_down_buf;

  // dummy beta, gamma variables
  // Variable beta_dummy, gamma_dummy;
  // const auto param_shape = this->mean_.shape();
  // if (this->no_bias_) {
  //   beta_dummy.reshape(param_shape, true);
  //   beta_dummy.data()->zero();
  // }
  if (this->no_scale_) {
    gamma_dummy_.data()->fill(1.);
  }

  // const void *beta =
  //     this->no_bias_
  //         ? beta_dummy.data()->get(DRV_BN_T(), this->ctx_)->const_pointer()
  //         : inputs[this->b_idx_]
  //               ->data()
  //               ->get(DRV_BN_T(), this->ctx_)
  //               ->const_pointer();

  const void *gamma =
      this->no_scale_
          ? gamma_dummy_.data()->get(DRV_BN_T(), this->ctx_)->const_pointer()
          : inputs[this->g_idx_]
                ->data()
                ->get(DRV_BN_T(), this->ctx_)
                ->const_pointer();

  // Specify write only flag to prevent unnecessary memset.
  const bool param_diff_write = b_param == 0;
  void *db = pd_beta
                 ? inputs[this->b_idx_]
                       ->grad()
                       ->cast(DRV_BN_T(), this->ctx_, param_diff_write)
                       ->pointer()
                 : prop_down_buf;
  void *dg = pd_gamma
                 ? inputs[this->g_idx_]
                       ->grad()
                       ->cast(DRV_BN_T(), this->ctx_, param_diff_write)
                       ->pointer()
                 : prop_down_buf;
  double eps = std::max((double)this->eps_, HIPDNN_BN_MIN_EPSILON);
  NBLA_CUDNN_CHECK(hipdnnBatchNormalizationBackward(
      cudnn_handle_, mode_, &a_data, &b_data, &a_param, &b_param,
      input_desc_.desc, x, output_desc_.desc, dy, input_desc_.desc, dx,
      bn_scale_bias_mean_var_desc_.desc, gamma, dg, db, eps, m, v));
}
}
