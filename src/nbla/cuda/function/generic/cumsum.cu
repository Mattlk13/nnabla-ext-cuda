#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/cumsum.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void CumSumCuda<T>::setup_impl(const Variables &inputs,
                               const Variables &outputs) {
  CumSum<T>::setup_impl(inputs, outputs);
  cuda_set_device(this->device_);
}

template <typename T, typename AccumType>
__global__ void kernel_cumsum_forward(const int size0x2_, const int size1_,
                                      const int size2_, const T *x,
                                      AccumType *y, bool exclusive_,
                                      bool reverse_) {
  NBLA_CUDA_KERNEL_LOOP(idx, size0x2_) {
    const int i0 = idx / size2_;
    const int i2 = idx % size2_;

    int j = i0 * size1_ * size2_ + i2;
    for (int index = 0; index < size1_; ++index) {
      const int i1 = reverse_ ? size1_ - index - 1 : index;

      const int d = reverse_ ? -1 : 1;
      const int x_k = exclusive_ ? (i1 - d) * size2_ + j : i1 * size2_ + j;
      const int y_k = i1 * size2_ + j;
      const int y_k_prev = y_k - d * size2_;

      y[y_k] = index != 0 ? y[y_k_prev] + x[x_k] : exclusive_ ? 0 : x[x_k];
    }
  }
}

template <typename T>
void CumSumCuda<T>::forward_impl(const Variables &inputs,
                                 const Variables &outputs) {
  cuda_set_device(this->device_);
  const Tcu *x = inputs[0]->get_data_pointer<Tcu>(this->ctx_);
  AccumType *y = outputs[0]->cast_data_and_get_pointer<Tcu>(this->ctx_, true);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
      kernel_cumsum_forward, this->size0_ * this->size2_, this->size1_,
      this->size2_, x, y, this->exclusive_, this->reverse_);
}

template <typename T>
__global__ void kernel_cumsum_backward(const int size0x2_, const int size1_,
                                       const int size2_, const T *g_y, T *g_x,
                                       bool exclusive_, bool reverse_,
                                       bool accum) {
  typedef typename CudaTypeForceFloat<T>::type AccumType;
  NBLA_CUDA_KERNEL_LOOP(idx, size0x2_) {
    const int i0 = idx / size2_;
    const int i2 = idx % size2_;
    const int j = i0 * size1_ * size2_ + i2;

    AccumType cum_sum = T(0);
    for (int index = 0; index < size1_; ++index) {

      const int i1 = reverse_ ? index : size1_ - index - 1;
      const int x_k = i1 * size2_ + j;

      cum_sum += g_y[x_k];
      if (accum)
        g_x[x_k] += (exclusive_ ? cum_sum - g_y[x_k] : cum_sum);
      else
        g_x[x_k] = (exclusive_ ? cum_sum - g_y[x_k] : cum_sum);
    }
  }
}

template <typename T>
void CumSumCuda<T>::backward_impl(const Variables &inputs,
                                  const Variables &outputs,
                                  const vector<bool> &propagate_down,
                                  const vector<bool> &accum) {
  if (!(propagate_down[0])) {
    return;
  }
  cuda_set_device(this->device_);

  const Tcu *g_y = outputs[0]->get_grad_pointer<Tcu>(this->ctx_);
  if (propagate_down[0]) {
    Tcu *g_x = inputs[0]->cast_grad_and_get_pointer<Tcu>(this->ctx_, !accum[0]);
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_cumsum_backward<Tcu>),
                                   this->size0_ * this->size2_, this->size1_,
                                   this->size2_, g_y, g_x, this->exclusive_,
                                   this->reverse_, accum[0]);
  }
}
}