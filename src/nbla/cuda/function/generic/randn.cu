// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/randn.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void RandnCuda<T>::setup_impl(const Variables &inputs,
                              const Variables &outputs) {
  Randn<T>::setup_impl(inputs, outputs);
}

template <typename T>
void RandnCuda<T>::forward_impl(const Variables &inputs,
                                const Variables &outputs) {
  typedef typename CudaTypeForceFloat<T>::type Tc;
  cuda_set_device(device_);
  hiprandGenerator_t &gen =
      this->seed_ == -1 ? SingletonManager::get<Cuda>()->curand_generator()
                        : curand_generator_;
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  curand_generate_randn<float>(gen, this->mu_, this->sigma_, y,
                               outputs[0]->size());
}

template <typename T>
void RandnCuda<T>::backward_impl(const Variables &inputs,
                                 const Variables &outputs,
                                 const vector<bool> &propagate_down,
                                 const vector<bool> &accum) {
  // Pass
}
}
