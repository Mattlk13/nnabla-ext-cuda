#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/function/sum.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/cuda/utils/reduce.cuh>

namespace nbla {

template <typename T>
void SumCuda<T>::forward_impl_reduce(const T *x_, T *y_, int outer_size,
                                     int reduction_size) {
  const Tc *x = reinterpret_cast<const Tc *>(x_);
  Tc *y = reinterpret_cast<Tc *>(y_);
  cuda_set_device(this->device_);

  if (reduction_size / outer_size < 2048) {
    const Tc *ones =
        static_cast<const Tc *>(SingletonManager::get<NNabla>()->ones(
            reduction_size, get_dtype<Tc>(), this->ctx_));
    cuda_gemv<Tc>(this->device_, y, x, reduction_size, outer_size, true, ones,
                  reduction_size, 1, 0);
  } else if (reduction_size >= 1024) {
    const int threads = NBLA_CUDA_NUM_THREADS;
    const int blocks = min(NBLA_CUDA_GET_BLOCKS(reduction_size), 1024);
    NdArray arr_buff({blocks});
    Tc *buff = arr_buff.cast(get_dtype<Tc>(), this->ctx_, true)->pointer<Tc>();
    while (outer_size--) {
      kernel_reduce_per_block<<<blocks, threads>>>(reduction_size, x, buff);
      NBLA_CUDA_KERNEL_CHECK();
      kernel_reduce_per_block<<<1, 1024>>>(blocks, buff, y);
      NBLA_CUDA_KERNEL_CHECK();
      x += reduction_size;
      y += 1;
    }
  } else {
    while (outer_size--) {
      kernel_reduce_per_block<<<1, 1024>>>(reduction_size, x, y);
      NBLA_CUDA_KERNEL_CHECK();
      x += reduction_size;
      y += 1;
    }
  }
}

template <typename T, bool accum>
__global__ void kernel_reduce_sum_backward(const int num, T *dx, const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { dx[idx] = (accum ? dx[idx] : (T)0) + *dy; }
}

template <typename T>
void SumCuda<T>::backward_impl_reduce(const T *dy_, T *dx_, int outer_size,
                                      int reduction_size, bool accum) {
  const Tc *dy = reinterpret_cast<const Tc *>(dy_);
  Tc *dx = reinterpret_cast<Tc *>(dx_);
  cuda_set_device(this->device_);
  if (outer_size == 1) {
    if (accum) {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_sum_backward<Tc, true>),
                                     reduction_size, dx, dy);
    } else {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_sum_backward<Tc, false>),
                                     reduction_size, dx, dy);
    }
    return;
  }
  const Tc *ones =
      static_cast<const Tc *>(SingletonManager::get<NNabla>()->ones(
          reduction_size, get_dtype<Tc>(), this->ctx_));
  cuda_gemm<Tc>(this->device_, dx, true, dy, outer_size, 1, false, ones, 1,
                reduction_size, false, 1, accum ? 1 : 0);
}
}
