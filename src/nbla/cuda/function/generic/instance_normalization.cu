#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/instance_normalization.hpp>
#include <nbla/variable.hpp>

// Common kernels and reduce ops
#include <nbla/cuda/function/kernel/normalization.cuh>

namespace nbla {

template <typename T>
void InstanceNormalizationCuda<T>::setup_impl(const Variables &inputs,
                                              const Variables &outputs) {
  InstanceNormalization<T>::setup_impl(inputs, outputs);
  cuda_set_device(this->device_);

  need_adaptor_ = ChannelFirstAdaptor::need_adaptor(
      inputs[0]->shape(), this->batch_axis_, this->channel_axis_);

  if (need_adaptor_) {
    adaptor_ = std::make_shared<ChannelFirstAdaptor>();
    adaptor_->setup(inputs[0], &pre_adaptor_, &post_adaptor_, outputs[0],
                    inputs[0]->shape(), this->batch_axis_, this->channel_axis_,
                    this->ctx_);

    reduce_size_ = pre_adaptor_.size(this->batch_axis_.size() + 1);
    outer_size_ = pre_adaptor_.size() / reduce_size_;
  } else {
    reduce_size_ = inputs[0]->size(this->channel_axis_ + 1);
    outer_size_ = inputs[0]->size() / reduce_size_;
  }

  mean_.reshape({outer_size_}, true);
  var_.reshape({outer_size_}, true);
  sum_dy_.reshape({outer_size_}, true);
  sum_dyx_.reshape({outer_size_}, true);
  factor_a_.reshape({outer_size_}, true);
  factor_b_.reshape({outer_size_}, true);
}

template <typename T, typename index_t>
__global__ void
instance_norm_forward_normalization(const index_t outer_size,
                                    const index_t reduce_size, const T *x,
                                    const T *mean, const T *var, const T *beta,
                                    const T *gamma, T *y, const float eps) {
  const index_t bidy = blockIdx.y;
  const index_t gdimy = gridDim.y;
  const index_t tidx = threadIdx.x;
  const index_t bdimx = blockDim.x;

  // Grid-stride loop
  for (index_t outer_idx = blockIdx.x; outer_idx < outer_size;
       outer_idx += gridDim.x) {
    for (index_t i = tidx + bdimx * bidy; i < reduce_size; i += bdimx * gdimy) {
      const index_t idx = outer_idx * reduce_size + i;
      const T scale = gamma ? gamma[outer_idx] : (T)1.0f;
      const T bias = beta ? beta[outer_idx] : (T)0.0f;
      const T invstd = rsqrt(var[outer_idx] + eps);

      y[idx] = scale * invstd * (x[idx] - mean[outer_idx]) + bias;
    }
  }
}

template <typename T, typename index_t>
__global__ void instance_norm_backward_dx_factor(
    const index_t outer_size, const index_t reduce_size, const T *gamma,
    const T *mean, const T *var, const T *dmean, const T *dvar, const T *sum_dy,
    const T *sum_dyx, T *factor_a, T *factor_b, const float eps) {
  // Grid-stride loop
  for (index_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < outer_size;
       idx += gridDim.x * blockDim.x) {
    const float inv_reduce_size = 1.0f / reduce_size;
    const float invstd = rsqrt(var[idx] + eps);
    const float scale = gamma ? static_cast<float>(gamma[idx]) : 1.0f;

    const float tmp = (sum_dy[idx] * scale * mean[idx] - sum_dyx[idx] * scale) *
                          invstd * invstd * invstd * inv_reduce_size +
                      (dvar ? 2.0f * dvar[idx] * inv_reduce_size : 0.0f);

    factor_a[idx] = tmp;
    factor_b[idx] = -tmp * mean[idx] -
                    sum_dy[idx] * scale * invstd * inv_reduce_size +
                    (dmean ? dmean[idx] * inv_reduce_size : 0.0f);
  }
}

template <bool accum, typename T, typename index_t>
__global__ void
instance_norm_backward_dx(const index_t outer_size, const index_t reduce_size,
                          const T *x, const T *gamma, const T *dy, const T *var,
                          const T *factor_a, const T *factor_b, T *dx,
                          const float eps) {

  const index_t bidy = blockIdx.y;
  const index_t gdimy = gridDim.y;
  const index_t tidx = threadIdx.x;
  const index_t bdimx = blockDim.x;

  // Grid-stride loop
  for (index_t outer_idx = blockIdx.x; outer_idx < outer_size;
       outer_idx += gridDim.x) {
    for (index_t i = tidx + bdimx * bidy; i < reduce_size; i += bdimx * gdimy) {
      const index_t idx = outer_idx * reduce_size + i;
      const T scale = gamma ? gamma[outer_idx] : (T)1.0f;
      const T invstd = rsqrt(var[outer_idx] + eps);

      if (accum) {
        dx[idx] += dy[idx] * invstd * scale + factor_a[outer_idx] * x[idx] +
                   factor_b[outer_idx];
      } else {
        dx[idx] = dy[idx] * invstd * scale + factor_a[outer_idx] * x[idx] +
                  factor_b[outer_idx];
      }
    }
  }
}

template <bool accum_beta, bool accum_gamma, typename T, typename index_t>
__global__ void instance_norm_backward_dbeta_dgamma(
    const index_t outer_size, const index_t reduce_size, const T *x,
    const T *gamma, const T *dy, const T *sum_dy, const T *sum_dyx,
    const T *mean, const T *var, T *dbeta_out, T *dgamma_out, const float eps) {
  // Grid-stride loop
  for (index_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < outer_size;
       idx += gridDim.x * blockDim.x) {
    const float invstd = rsqrt(var[idx] + eps);
    const float dbeta = sum_dy[idx];
    const float dgamma =
        sum_dyx[idx] * invstd - sum_dy[idx] * mean[idx] * invstd;

    if (dbeta_out) {
      if (accum_beta) {
        dbeta_out[idx] += dbeta;
      } else {
        dbeta_out[idx] = dbeta;
      }
    }
    if (dgamma_out) {
      if (accum_gamma) {
        dgamma_out[idx] += dgamma;
      } else {
        dgamma_out[idx] = dgamma;
      }
    }
  }
}

template <typename T>
void InstanceNormalizationCuda<T>::forward_impl(const Variables &inputs,
                                                const Variables &outputs) {
  cuda_set_device(this->device_);
  if (need_adaptor_) {
    // Transpose input to [B, C, H, W] memory format.
    adaptor_->forward_pre(inputs[0], &pre_adaptor_);

    auto in_cf_in = inputs;
    auto in_cf_out = outputs;
    in_cf_in[0] = &pre_adaptor_;
    in_cf_out[0] = &post_adaptor_;

    // Instance normalization
    forward_channel_first(in_cf_in, in_cf_out);

    // Transpose output to original memory format.
    adaptor_->forward_post(&post_adaptor_, outputs[0]);
  } else {
    forward_channel_first(inputs, outputs);
  }
}

template <typename T>
void InstanceNormalizationCuda<T>::forward_channel_first(
    const Variables &inputs, const Variables &outputs) {
  cuda_set_device(this->device_);

  Variable *v_mean = &mean_;
  Variable *v_var = &var_;
  // Output mean and var when output_stats == true.
  if (outputs.size() == 3) {
    v_mean = outputs[1];
    v_var = outputs[2];
  }

  // Calculate mean and variance
  {
    const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
    Tc *mean = v_mean->cast_data_and_get_pointer<Tc>(this->ctx_);
    Tc *var = v_var->cast_data_and_get_pointer<Tc>(this->ctx_);

    const int num_threads =
        reduce_size_ < IN_NUM_THREADS ? CUDA_WARP_SIZE : IN_NUM_THREADS;

    const auto grid = std::min(outer_size_, static_cast<Size_t>(IN_MAX_BLOCKS));
    const auto block = num_threads;

    WelfordOp<Tc, Size_t> op(x, mean, var, reduce_size_);
    reduce_2d_x<<<grid, block>>>(op, outer_size_, reduce_size_);
    NBLA_CUDA_KERNEL_CHECK();
  }

  // Instance normalization
  {
    const auto beta_idx = 1;
    const auto gamma_idx = this->no_bias_ ? 1 : 2;

    const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
    const Tc *mean = v_mean->get_data_pointer<Tc>(this->ctx_);
    const Tc *var = v_var->get_data_pointer<Tc>(this->ctx_);
    const Tc *beta = this->no_bias_
                         ? nullptr
                         : inputs[beta_idx]->get_data_pointer<Tc>(this->ctx_);
    const Tc *gamma = this->no_scale_
                          ? nullptr
                          : inputs[gamma_idx]->get_data_pointer<Tc>(this->ctx_);
    Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_);

    const size_t elements_per_grid_y = IN_NUM_THREADS * 4;
    dim3 grid;
    grid.x = std::min(outer_size_, static_cast<Size_t>(IN_MAX_BLOCKS));
    grid.y = std::min(NBLA_CEIL_SIZE_T_DIV(reduce_size_, elements_per_grid_y),
                      static_cast<Size_t>(IN_MAX_BLOCKS));
    grid.z = 1;
    const auto block = IN_NUM_THREADS;

    instance_norm_forward_normalization<<<grid, block>>>(
        outer_size_, reduce_size_, x, mean, var, beta, gamma, y, this->eps_);
    NBLA_CUDA_KERNEL_CHECK();
  }
}

template <typename T>
void InstanceNormalizationCuda<T>::backward_impl(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  if (!(propagate_down[0] || (inputs.size() > 1 && propagate_down[1]) ||
        (inputs.size() > 2 && propagate_down[2]))) {
    return;
  }
  cuda_set_device(this->device_);

  if (need_adaptor_) {
    adaptor_->backward_post(&post_adaptor_, outputs[0], true, false);

    auto in_cf_in = inputs;
    auto in_cf_out = outputs;
    in_cf_in[0] = &pre_adaptor_;
    in_cf_out[0] = &post_adaptor_;

    auto in_cf_accum = accum;
    in_cf_accum[0] = false;
    backward_channel_first(in_cf_in, in_cf_out, propagate_down, in_cf_accum);

    adaptor_->backward_pre(inputs[0], &pre_adaptor_, propagate_down[0],
                           accum[0]);
  } else {
    backward_channel_first(inputs, outputs, propagate_down, accum);
  }
}

template <typename T>
void InstanceNormalizationCuda<T>::backward_channel_first(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  Variable *v_mean = &mean_;
  Variable *v_var = &var_;
  // Output mean and var when output_stats == true.
  if (outputs.size() == 3) {
    v_mean = outputs[1];
    v_var = outputs[2];
  }

  // Calculate sum of dy and sum of dy * x.
  {
    const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
    const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
    Tc *sum_dy = sum_dy_.cast_data_and_get_pointer<Tc>(this->ctx_);
    Tc *sum_dyx = sum_dyx_.cast_data_and_get_pointer<Tc>(this->ctx_);

    const int num_threads =
        reduce_size_ < IN_NUM_THREADS ? CUDA_WARP_SIZE : IN_NUM_THREADS;

    const auto grid = std::min(outer_size_, static_cast<Size_t>(IN_MAX_BLOCKS));

    const auto block = num_threads;

    INGradOp<Tc, Size_t> op(x, dy, sum_dy, sum_dyx);
    reduce_2d_x<<<grid, block>>>(op, outer_size_, reduce_size_);
    NBLA_CUDA_KERNEL_CHECK();
  }

  // TODO: change the comment
  // Calculate a and b such that `dx = gamma / sqrt(var) * dy + a * x + b`.
  {
    const auto gamma_idx = this->no_bias_ ? 1 : 2;
    const Tc *gamma = this->no_scale_
                          ? nullptr
                          : inputs[gamma_idx]->get_data_pointer<Tc>(this->ctx_);
    const Tc *mean = v_mean->get_data_pointer<Tc>(this->ctx_);
    const Tc *var = v_var->get_data_pointer<Tc>(this->ctx_);
    const Tc *dmean = outputs.size() == 3
                          ? v_mean->get_grad_pointer<Tc>(this->ctx_)
                          : nullptr;
    const Tc *dvar =
        outputs.size() == 3 ? v_var->get_grad_pointer<Tc>(this->ctx_) : nullptr;
    const Tc *sum_dy = sum_dy_.get_data_pointer<Tc>(this->ctx_);
    const Tc *sum_dyx = sum_dyx_.get_data_pointer<Tc>(this->ctx_);

    Tc *factor_a = factor_a_.cast_data_and_get_pointer<Tc>(this->ctx_);
    Tc *factor_b = factor_b_.cast_data_and_get_pointer<Tc>(this->ctx_);

    const auto grid = std::min(
        static_cast<Size_t>(IN_MAX_BLOCKS),
        static_cast<Size_t>(NBLA_CEIL_SIZE_T_DIV(outer_size_, IN_NUM_THREADS)));
    const auto block = IN_NUM_THREADS;

    instance_norm_backward_dx_factor<<<grid, block>>>(
        outer_size_, reduce_size_, gamma, mean, var, dmean, dvar, sum_dy,
        sum_dyx, factor_a, factor_b, this->eps_);
    NBLA_CUDA_KERNEL_CHECK();
  }

  // Calculate dx.
  if (propagate_down[0]) {
    const auto gamma_idx = this->no_bias_ ? 1 : 2;

    const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
    const Tc *gamma = this->no_scale_
                          ? nullptr
                          : inputs[gamma_idx]->get_data_pointer<Tc>(this->ctx_);
    const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
    const Tc *var = v_var->get_data_pointer<Tc>(this->ctx_);
    const Tc *factor_a = factor_a_.get_data_pointer<Tc>(this->ctx_);
    const Tc *factor_b = factor_b_.get_data_pointer<Tc>(this->ctx_);

    Tc *dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, !accum[0]);

    const size_t elements_per_grid_y = IN_NUM_THREADS * 4;
    dim3 grid;
    grid.x = std::min(outer_size_, static_cast<Size_t>(IN_MAX_BLOCKS));
    grid.y = std::min(NBLA_CEIL_SIZE_T_DIV(reduce_size_, elements_per_grid_y),
                      static_cast<Size_t>(IN_MAX_BLOCKS));
    grid.z = 1;
    const auto block = IN_NUM_THREADS;

    if (accum[0]) {
      instance_norm_backward_dx<true><<<grid, block>>>(
          outer_size_, reduce_size_, x, gamma, dy, var, factor_a, factor_b, dx,
          this->eps_);
    } else {
      instance_norm_backward_dx<false><<<grid, block>>>(
          outer_size_, reduce_size_, x, gamma, dy, var, factor_a, factor_b, dx,
          this->eps_);
    }
    NBLA_CUDA_KERNEL_CHECK();
  }

  // Calculate dbeta and dgamma.
  if ((inputs.size() > 1 && propagate_down[1]) ||
      (inputs.size() > 2 && propagate_down[2])) {
    const auto beta_idx = 1;
    const auto gamma_idx = this->no_bias_ ? 1 : 2;

    const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
    const Tc *gamma = this->no_scale_
                          ? nullptr
                          : inputs[gamma_idx]->get_data_pointer<Tc>(this->ctx_);
    const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
    const Tc *sum_dy = sum_dy_.get_data_pointer<Tc>(this->ctx_);
    const Tc *sum_dyx = sum_dyx_.get_data_pointer<Tc>(this->ctx_);
    const Tc *mean = v_mean->get_data_pointer<Tc>(this->ctx_);
    const Tc *var = v_var->get_data_pointer<Tc>(this->ctx_);
    Tc *dbeta = !this->no_bias_ && propagate_down[beta_idx]
                    ? inputs[beta_idx]->cast_grad_and_get_pointer<Tc>(
                          this->ctx_, !accum[beta_idx])
                    : nullptr;
    Tc *dgamma = !this->no_scale_ && propagate_down[gamma_idx]
                     ? inputs[gamma_idx]->cast_grad_and_get_pointer<Tc>(
                           this->ctx_, !accum[gamma_idx])
                     : nullptr;

    const auto grid = std::min(
        static_cast<Size_t>(IN_MAX_BLOCKS),
        static_cast<Size_t>(NBLA_CEIL_SIZE_T_DIV(outer_size_, IN_NUM_THREADS)));
    const auto block = IN_NUM_THREADS;

    if (!this->no_bias_ && accum[beta_idx]) {
      if (!this->no_scale_ && accum[gamma_idx]) {
        instance_norm_backward_dbeta_dgamma<true, true><<<grid, block>>>(
            outer_size_, reduce_size_, x, gamma, dy, sum_dy, sum_dyx, mean, var,
            dbeta, dgamma, this->eps_);
      } else {
        instance_norm_backward_dbeta_dgamma<true, false><<<grid, block>>>(
            outer_size_, reduce_size_, x, gamma, dy, sum_dy, sum_dyx, mean, var,
            dbeta, dgamma, this->eps_);
      }
    } else {
      if (!this->no_scale_ && accum[gamma_idx]) {
        instance_norm_backward_dbeta_dgamma<false, true><<<grid, block>>>(
            outer_size_, reduce_size_, x, gamma, dy, sum_dy, sum_dyx, mean, var,
            dbeta, dgamma, this->eps_);
      } else {
        instance_norm_backward_dbeta_dgamma<false, false><<<grid, block>>>(
            outer_size_, reduce_size_, x, gamma, dy, sum_dy, sum_dyx, mean, var,
            dbeta, dgamma, this->eps_);
      }
    }
    NBLA_CUDA_KERNEL_CHECK();
  }
}
}
